#include "hip/hip_runtime.h"
/*
 *  gProximity Library.
 *  
 *  
 *  Copyright (C) 2010 University of North Carolina at Chapel Hill.
 *  All rights reserved.
 *  
 *  Permission to use, copy, modify, and distribute this software and its
 *  documentation for educational, research, and non-profit purposes, without
 *  fee, and without a written agreement is hereby granted, provided that the
 *  above copyright notice, this paragraph, and the following four paragraphs
 *  appear in all copies.
 *  
 *  Permission to incorporate this software into commercial products may be
 *  obtained by contacting the University of North Carolina at Chapel Hill.
 *  
 *  This software program and documentation are copyrighted by the University of
 *  North Carolina at Chapel Hill. The software program and documentation are
 *  supplied "as is", without any accompanying services from the University of
 *  North Carolina at Chapel Hill or the authors. The University of North
 *  Carolina at Chapel Hill and the authors do not warrant that the operation of
 *  the program will be uninterrupted or error-free. The end-user understands
 *  that the program was developed for research purposes and is advised not to
 *  rely exclusively on the program for any reason.
 *  
 *  IN NO EVENT SHALL THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL OR ITS
 *  EMPLOYEES OR THE AUTHORS BE LIABLE TO ANY PARTY FOR DIRECT, INDIRECT,
 *  SPECIAL, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, INCLUDING LOST PROFITS,
 *  ARISING OUT OF THE USE OF THIS SOFTWARE AND ITS DOCUMENTATION, EVEN IF THE
 *  UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL OR THE AUTHORS HAVE BEEN ADVISED
 *  OF THE POSSIBILITY OF SUCH DAMAGE.
 *  
 *  THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL AND THE AUTHORS SPECIFICALLY
 *  DISCLAIM ANY WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 *  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE AND ANY
 *  STATUTORY WARRANTY OF NON-INFRINGEMENT. THE SOFTWARE PROVIDED HEREUNDER IS
 *  ON AN "AS IS" BASIS, AND THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL AND
 *  THE AUTHORS HAVE NO OBLIGATIONS TO PROVIDE MAINTENANCE, SUPPORT, UPDATES,
 *  ENHANCEMENTS, OR MODIFICATIONS.
 *  
 *  Please send all BUG REPORTS to:
 *  
 *  geom@cs.unc.edu
 *  
 *  The authors may be contacted via:
 *  
 *  Christian Lauterbach, Qi Mo, Jia Pan and Dinesh Manocha
 *  Dept. of Computer Science
 *  Frederick P. Brooks Jr. Computer Science Bldg.
 *  3175 University of N.C.
 *  Chapel Hill, N.C. 27599-3175
 *  United States of America
 *  
 *  http://gamma.cs.unc.edu/GPUCOL/
 *  
 */
 
{
#ifndef UPDATE_TRIARRAY
#define UPDATE_TRIARRAY BVH.triIDs
#endif

	//
	// Updates a bounding box on multiple threads in parallel.
	// Output is stored in the first index of localBoxes, which is assumed to
	// be a shared memory array of 6*nThreads floats.
	//
	
	// NB: localBoxes is arranged as an struct of arrays instead of array of structs,
	// i.e. all the min[0] values are stored first, then all min[1] etc.
	// This ensures shared memory accesses are conflict free.
	
	// if the number of boxes is pretty low (compared to #threads), then
	// the hierarchical reduction should only be run on as many threads
	// as are necessary, so we compute the next largest power of two for
	// the number of boxes here to detect this.
	int nextPowerTwo = nBoxesToProcess;
	{
		nextPowerTwo--;
		nextPowerTwo |= nextPowerTwo >> 1;
		nextPowerTwo |= nextPowerTwo >> 2;
		nextPowerTwo |= nextPowerTwo >> 4;
		nextPowerTwo |= nextPowerTwo >> 8;
		nextPowerTwo |= nextPowerTwo >> 16;
		nextPowerTwo++;
		nextPowerTwo = min(nextPowerTwo, nSplitThreads);
	}
	
	// go through the list of boxes in chunks, each thread reading one of the boxes and
	// building the bounding box locally
	{
#ifndef UPDATE_EXTBBOX
		AABB localBox; // AABB in registers, to be updated by each thread
		AABB_INIT(localBox);
#define UPDATE_EXTBOX localBox
#endif
		while(nBoxesToProcess > 0)
		{
			if(threadOffset < nBoxesToProcess)
			{
				const int triID = UPDATE_TRIARRAY[boxOffset + threadOffset];
				float2 box;
				
				// X
				box = tex1Dfetch(g_tex_AABBs, triID);
				UPDATE_EXTBOX.bb_min.x = min(UPDATE_EXTBOX.bb_min.x, box.x);
				UPDATE_EXTBOX.bb_max.x = max(UPDATE_EXTBOX.bb_max.x, box.y);
				// Y
				box = tex1Dfetch(g_tex_AABBs, triID + g_tex_AABBs_offset);
				UPDATE_EXTBOX.bb_min.y = min(UPDATE_EXTBOX.bb_min.y, box.x);
				UPDATE_EXTBOX.bb_max.y = max(UPDATE_EXTBOX.bb_max.y, box.y);
				// Z
				box = tex1Dfetch(g_tex_AABBs, triID + 2 * g_tex_AABBs_offset);
				UPDATE_EXTBOX.bb_min.z = min(UPDATE_EXTBOX.bb_min.z, box.x);
				UPDATE_EXTBOX.bb_max.z = max(UPDATE_EXTBOX.bb_max.z, box.y);
			}
			
			boxOffset += nSplitThreads;
			nBoxesToProcess -= nSplitThreads;
		}
		
		if(threadOffset < nextPowerTwo)
		{
			// save accumulated bounding boxes to shared memory
			localBoxes[threadOffset]				   = UPDATE_EXTBOX.bb_min.x;
			localBoxes[threadOffset + nSplitThreads]   = UPDATE_EXTBOX.bb_min.y;
			localBoxes[threadOffset + 2*nSplitThreads] = UPDATE_EXTBOX.bb_min.z;
			localBoxes[threadOffset + 3*nSplitThreads] = UPDATE_EXTBOX.bb_max.x;
			localBoxes[threadOffset + 4*nSplitThreads] = UPDATE_EXTBOX.bb_max.y;
			localBoxes[threadOffset + 5*nSplitThreads] = UPDATE_EXTBOX.bb_max.z;
		}
		__syncthreads();
	}
	
	// merge bounding boxes by parallel reduce:
	for(int r = nextPowerTwo / 2; r != 0; r /= 2)
	{
		if(threadOffset < r)
		{
			int other = threadOffset + r;
			
			// min[0]..min[2]
			localBoxes[threadOffset                  ] = min(localBoxes[threadOffset                  ], localBoxes[other                  ]);
			localBoxes[threadOffset + nSplitThreads  ] = min(localBoxes[threadOffset + nSplitThreads  ], localBoxes[other + nSplitThreads  ]);
			localBoxes[threadOffset + 2*nSplitThreads] = min(localBoxes[threadOffset + 2*nSplitThreads], localBoxes[other + 2*nSplitThreads]);
			
			// max[0]..max[2]
			localBoxes[threadOffset + 3*nSplitThreads] = max(localBoxes[threadOffset + 3*nSplitThreads], localBoxes[other + 3*nSplitThreads]);
			localBoxes[threadOffset + 4*nSplitThreads] = max(localBoxes[threadOffset + 4*nSplitThreads], localBoxes[other + 4*nSplitThreads]);
			localBoxes[threadOffset + 5*nSplitThreads] = max(localBoxes[threadOffset + 5*nSplitThreads], localBoxes[other + 5*nSplitThreads]);
		}
		
		__syncthreads();
	}
	
#undef UPDATE_TRIARRAY
#undef UPDATE_EXTBBOX
}
