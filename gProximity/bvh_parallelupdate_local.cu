#include "hip/hip_runtime.h"
/*
 *  gProximity Library.
 *  
 *  
 *  Copyright (C) 2010 University of North Carolina at Chapel Hill.
 *  All rights reserved.
 *  
 *  Permission to use, copy, modify, and distribute this software and its
 *  documentation for educational, research, and non-profit purposes, without
 *  fee, and without a written agreement is hereby granted, provided that the
 *  above copyright notice, this paragraph, and the following four paragraphs
 *  appear in all copies.
 *  
 *  Permission to incorporate this software into commercial products may be
 *  obtained by contacting the University of North Carolina at Chapel Hill.
 *  
 *  This software program and documentation are copyrighted by the University of
 *  North Carolina at Chapel Hill. The software program and documentation are
 *  supplied "as is", without any accompanying services from the University of
 *  North Carolina at Chapel Hill or the authors. The University of North
 *  Carolina at Chapel Hill and the authors do not warrant that the operation of
 *  the program will be uninterrupted or error-free. The end-user understands
 *  that the program was developed for research purposes and is advised not to
 *  rely exclusively on the program for any reason.
 *  
 *  IN NO EVENT SHALL THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL OR ITS
 *  EMPLOYEES OR THE AUTHORS BE LIABLE TO ANY PARTY FOR DIRECT, INDIRECT,
 *  SPECIAL, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, INCLUDING LOST PROFITS,
 *  ARISING OUT OF THE USE OF THIS SOFTWARE AND ITS DOCUMENTATION, EVEN IF THE
 *  UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL OR THE AUTHORS HAVE BEEN ADVISED
 *  OF THE POSSIBILITY OF SUCH DAMAGE.
 *  
 *  THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL AND THE AUTHORS SPECIFICALLY
 *  DISCLAIM ANY WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 *  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE AND ANY
 *  STATUTORY WARRANTY OF NON-INFRINGEMENT. THE SOFTWARE PROVIDED HEREUNDER IS
 *  ON AN "AS IS" BASIS, AND THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL AND
 *  THE AUTHORS HAVE NO OBLIGATIONS TO PROVIDE MAINTENANCE, SUPPORT, UPDATES,
 *  ENHANCEMENTS, OR MODIFICATIONS.
 *  
 *  Please send all BUG REPORTS to:
 *  
 *  geom@cs.unc.edu
 *  
 *  The authors may be contacted via:
 *  
 *  Christian Lauterbach, Qi Mo, Jia Pan and Dinesh Manocha
 *  Dept. of Computer Science
 *  Frederick P. Brooks Jr. Computer Science Bldg.
 *  3175 University of N.C.
 *  Chapel Hill, N.C. 27599-3175
 *  United States of America
 *  
 *  http://gamma.cs.unc.edu/GPUCOL/
 *  
 */
 
{
	//
	// Updates a bounding box on multiple threads in parallel.
	// Output is stored in the first index of localBoxes, which is assumed to
	// be a shared memory array of 6*nThreads floats.
	//
	
	// NB: localBoxes is arranged as an struct of arrays instead of array of structs,
	// i.e. all the min[0] values are stored first, then all min[1] etc.
	// This ensures shared memory accesses are conflict free.
	
	
	if(threadOffset < nBoxesToProcess)
	{
		const int triID = localTriIDs[boxOffset + threadOffset];
		
		localBoxes[threadOffset]				   = localBoundingBoxes[triID];
		localBoxes[threadOffset +   nSplitThreads] = localBoundingBoxes[triID +   nMaxTris];
		localBoxes[threadOffset + 2*nSplitThreads] = localBoundingBoxes[triID + 2*nMaxTris];
		localBoxes[threadOffset + 3*nSplitThreads] = localBoundingBoxes[triID + 3*nMaxTris];
		localBoxes[threadOffset + 4*nSplitThreads] = localBoundingBoxes[triID + 4*nMaxTris];
		localBoxes[threadOffset + 5*nSplitThreads] = localBoundingBoxes[triID + 5*nMaxTris];
	}
	
	__syncthreads();
	
#define MERGE_NAIVE_THRESHOLD 4
	if(nBoxesToProcess <= MERGE_NAIVE_THRESHOLD)
	{
		if(threadOffset < 3)
		{
			int myOffset1 = threadOffset * nSplitThreads;
			int myOffset2 = (threadOffset + 3) * nSplitThreads;
			float myVal1 = localBoxes[myOffset1];
			float myVal2 = localBoxes[myOffset2];
			for(int i = 1; i < nBoxesToProcess; i++)
			{
				myVal1 = min(myVal1, localBoxes[myOffset1 + i]);
				myVal2 = max(myVal2, localBoxes[myOffset2 + i]);
			}
			localBoxes[threadOffset] = myVal1;
			localBoxes[threadOffset + 3] = myVal2;
		}
		__syncthreads();
	}
	else   // merge bounding boxes by parallel reduce:
	{
		for(int r = nSplitThreads / 2; r != 0; r /= 2)
		{
			int other = threadOffset + r;
			
			if(threadOffset < r && other < nBoxesToProcess)
			{
			
				// min[0]..min[2]
				localBoxes[threadOffset                  ] = min(localBoxes[threadOffset                  ], localBoxes[other                  ]);
				localBoxes[threadOffset + nSplitThreads  ] = min(localBoxes[threadOffset + nSplitThreads  ], localBoxes[other + nSplitThreads  ]);
				localBoxes[threadOffset + 2*nSplitThreads] = min(localBoxes[threadOffset + 2*nSplitThreads], localBoxes[other + 2*nSplitThreads]);
				
				// max[0]..max[2]
				localBoxes[threadOffset + 3*nSplitThreads] = max(localBoxes[threadOffset + 3*nSplitThreads], localBoxes[other + 3*nSplitThreads]);
				localBoxes[threadOffset + 4*nSplitThreads] = max(localBoxes[threadOffset + 4*nSplitThreads], localBoxes[other + 4*nSplitThreads]);
				localBoxes[threadOffset + 5*nSplitThreads] = max(localBoxes[threadOffset + 5*nSplitThreads], localBoxes[other + 5*nSplitThreads]);
			}
			
			__syncthreads();
		}
		
		if(threadOffset < 6)
			localBoxes[threadOffset] = localBoxes[threadOffset*nSplitThreads];
		__syncthreads();
	}
}
