#include "hip/hip_runtime.h"
/*
 *  gProximity Library.
 *  
 *  
 *  Copyright (C) 2010 University of North Carolina at Chapel Hill.
 *  All rights reserved.
 *  
 *  Permission to use, copy, modify, and distribute this software and its
 *  documentation for educational, research, and non-profit purposes, without
 *  fee, and without a written agreement is hereby granted, provided that the
 *  above copyright notice, this paragraph, and the following four paragraphs
 *  appear in all copies.
 *  
 *  Permission to incorporate this software into commercial products may be
 *  obtained by contacting the University of North Carolina at Chapel Hill.
 *  
 *  This software program and documentation are copyrighted by the University of
 *  North Carolina at Chapel Hill. The software program and documentation are
 *  supplied "as is", without any accompanying services from the University of
 *  North Carolina at Chapel Hill or the authors. The University of North
 *  Carolina at Chapel Hill and the authors do not warrant that the operation of
 *  the program will be uninterrupted or error-free. The end-user understands
 *  that the program was developed for research purposes and is advised not to
 *  rely exclusively on the program for any reason.
 *  
 *  IN NO EVENT SHALL THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL OR ITS
 *  EMPLOYEES OR THE AUTHORS BE LIABLE TO ANY PARTY FOR DIRECT, INDIRECT,
 *  SPECIAL, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, INCLUDING LOST PROFITS,
 *  ARISING OUT OF THE USE OF THIS SOFTWARE AND ITS DOCUMENTATION, EVEN IF THE
 *  UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL OR THE AUTHORS HAVE BEEN ADVISED
 *  OF THE POSSIBILITY OF SUCH DAMAGE.
 *  
 *  THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL AND THE AUTHORS SPECIFICALLY
 *  DISCLAIM ANY WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 *  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE AND ANY
 *  STATUTORY WARRANTY OF NON-INFRINGEMENT. THE SOFTWARE PROVIDED HEREUNDER IS
 *  ON AN "AS IS" BASIS, AND THE UNIVERSITY OF NORTH CAROLINA AT CHAPEL HILL AND
 *  THE AUTHORS HAVE NO OBLIGATIONS TO PROVIDE MAINTENANCE, SUPPORT, UPDATES,
 *  ENHANCEMENTS, OR MODIFICATIONS.
 *  
 *  Please send all BUG REPORTS to:
 *  
 *  geom@cs.unc.edu
 *  
 *  The authors may be contacted via:
 *  
 *  Christian Lauterbach, Qi Mo, Jia Pan and Dinesh Manocha
 *  Dept. of Computer Science
 *  Frederick P. Brooks Jr. Computer Science Bldg.
 *  3175 University of N.C.
 *  Chapel Hill, N.C. 27599-3175
 *  United States of America
 *  
 *  http://gamma.cs.unc.edu/GPUCOL/
 *  
 */
 
#include "bvh_kernels.h"


__global__ void generateAABBsIndexed(float2 *boxes, int *triIDs, const GPUVertex *vertices, uint3* triangles, unsigned int *zCode,
                                     const unsigned int nTris, const unsigned int array_offset, const float3 sceneBB_min,
                                     const float3 sceneBB_max, const int nVerts)
{
	int triID = blockIdx.x * blockDim.x + threadIdx.x;
	float3 sceneDim = f3v_sub(sceneBB_max, sceneBB_min);
	//const float lim = float(UINT_MAX>>(32-10));
	const float lim = float(1 << LBVH_GRID_LEVELS);
	sceneDim.x = lim / sceneDim.x;
	sceneDim.y = lim / sceneDim.y;
	sceneDim.z = lim / sceneDim.z;
	
	// only when still within triangle bounds
	if(triID < nTris)
	{
		// start new AABB
		AABB newBB;
		uint3 triIndices = triangles[triID];
		float3 v = vertices[triIndices.x].v;
		newBB.bb_min = v;
		newBB.bb_max = v;
		
		v = vertices[triIndices.y].v;
		AABB_UPDATE(newBB, v);
		
		v = vertices[triIndices.z].v;
		AABB_UPDATE(newBB, v);
		
		// centroid of box
		float3 centroid = f3v_add(newBB.bb_min, newBB.bb_max);
		centroid = f3v_mul1(centroid, 0.5f);
		
		// quantize from scene BB
		centroid = f3v_sub(centroid, sceneBB_min);
		
		// quantized coordinate in scene BB
		int3 code3;
		code3.x = floor(centroid.x * sceneDim.x);
		code3.y = floor(centroid.y * sceneDim.y);
		code3.z = floor(centroid.z * sceneDim.z);
		
		// z code
		int code = 0;
		code3.x = code3.x << 2;
		code3.y = code3.y << 1;
		for(int j = 9; j >= 0; j--)
		{
			unsigned int triple = ((code3.x >> j) & 0x4)
			                      | ((code3.y >> j) & 0x2)
			                      | ((code3.z >> j) & 0x1);
			                      
			code = (code << 3) | triple;
		}
		
		if(zCode)
			zCode[triID] = code;
			
		// store AABB in 1-D array
		boxes[triID]		  = make_float2(newBB.bb_min.x, newBB.bb_max.x); // X
		unsigned int offset   = array_offset;
		boxes[offset + triID] = make_float2(newBB.bb_min.y, newBB.bb_max.y); // Y
		offset               += array_offset;
		boxes[offset + triID] = make_float2(newBB.bb_min.z, newBB.bb_max.z); // Z
		
		// store triID in linear array
		triIDs[triID] = triID;
	}
}

__global__ void AABBtoOBBbyLevel(AABBNode *treeIn, OBBNode *treeOut, unsigned int startOffset, unsigned int nNodes, const GPUVertex *vertices, const uint3 *triangles, const int vertexOffset)
{
	const int threadOffset = blockIdx.x * blockDim.x + threadIdx.x;
	const int myOffset = startOffset + threadOffset;
	
	if(threadOffset < nNodes)
	{
		// read in node child pointer
		unsigned int childPtr = treeIn[myOffset].left;
		treeOut[myOffset].left = childPtr;
		unsigned int childPtr2 = treeIn[myOffset].right;
		treeOut[myOffset].right = childPtr2;
		
		// if leaf, update from triangles
		if((childPtr & 3) == 3)
		{
			uint3 idx = triangles[childPtr >> 2];
			float3 v1 = vertices[idx.x].v;
			float3 v2 = vertices[idx.y].v;
			float3 v3 = vertices[idx.z].v;
			
			
			treeOut[myOffset].bbox.init(v1, v2, v3);
		}
		else   // otherwise, update from child nodes
		{
			unsigned int leftChild = myOffset + (childPtr >> 5);
			OBB bv1 = treeOut[leftChild].bbox;
			OBB bv2 = treeOut[leftChild+1].bbox;
			treeOut[myOffset].bbox.init(bv1, bv2);
		}
	}
}

__global__ void copyGPUVertex(GPUVertex* v1, float* v2, unsigned int nVertices, unsigned int nTargetVertices, bool useMin, unsigned int d)
{
	unsigned int threadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	if(threadId >= nTargetVertices)
		return;

	if(threadId >= nVertices)
	{
		v2[threadId] = useMin ? -HIP_NORM_HUGE_F : HIP_NORM_HUGE_F;
	}
	else
	{
		switch(d)
		{
		case 0:
			{
				v2[threadId] = v1[threadId].v.x;
			}
			break;
		case 1:
			{
				v2[threadId] = v1[threadId].v.y;
			}
			break;
		case 2:
			{
				v2[threadId] = v1[threadId].v.z;
			}
			break;
		}
	}
}